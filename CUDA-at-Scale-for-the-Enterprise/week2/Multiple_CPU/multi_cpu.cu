#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Vector multiplication: C = A * B.
 *
 * This sample is a very basic sample that implements element by element
 * vector multiplication. It is based on the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include "multi_cpu.h"

/*
 * CUDA Kernel Device code
 *
 * Determines whether each item in a is  > = < to each element in c.
 *  If a[i] > b[i] => 1
 *  If a[i] = b[i] => 0
 *  If a[i] < b[i] => -1
 * You can use conditional branching or something similar but you may be able to use math operations to minimize the branching costs
 */
__global__ void compare(const int *a, const int *b, int *c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        // This is where the game is played
        if(a[i] > b[i]) c[i] = 1;
        else if(a[i] < b[i]) c[i] = -1;
        else c[i] = 0;
    }
}

__host__ std::tuple<int *, int *, int *> allocateHostMemory(int numElements)
{
    size_t size = numElements * sizeof(int);

    // Allocate the host input vector a
    int *h_a = (int *)malloc(size);

    // Allocate the host input vector b
    int *h_b = (int *)malloc(size);

    // Allocate the host output vector c
    int *h_c;
    hipMallocManaged((int **)&h_c, size);


    // Verify that allocations succeeded
    if (h_a == NULL || h_b == NULL || h_c == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    return {h_a, h_b, h_c};
}

__host__ std::tuple<int *, int *> allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector a
    int *d_a = NULL;
    size_t size = numElements * sizeof(int);
    hipError_t err = hipMalloc(&d_a, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector b
    int *d_b = NULL;
    err = hipMalloc(&d_b, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return {d_a, d_b};
}

__host__ void copyFromHostToDevice(int *hos, int *dev, int numElements)
{
    size_t size = numElements * sizeof(int);
    // Copy the host input vector to the device input vectors
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpy(dev, hos, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void executeKernel(int *d_a, int *d_b, int *c, int numElements)
{
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // REPLACE x, y, z with a, b, and c variables for memory on the GPU
    compare<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, c, numElements);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


// Free device global memory
__host__ void deallocateMemory(int *h_a, int *h_b, int *h_c, int *d_a, int *d_b)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipFree(d_a);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_b);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Based on http://www.cplusplus.com/forum/general/170845/
__host__ void placeDataToFiles(int *h_c, int numElements)
{
    printf("Placing calculation results into output files\n");
    ofstream outfile_a;
	outfile_a.open("./output_a.csv");
    ofstream outfile_b;
	outfile_b.open("./output_b.csv");

    // print first column's element
    outfile_a << h_c[0];
    outfile_b << "-" << h_c[0];

    for (int i=1; i < numElements; i++)
    {
        outfile_a << "," << h_c[i];
        outfile_b << "," << -(h_c[i]);
    }

    outfile_a << endl;
    outfile_b << endl;
}

// Based on content found at https://www.cplusplus.com/reference/cstdio
__host__ void retrieveDataFromFiles(int *h_a, int *h_b, int numElements)
{
    printf("Retrieving data from input files.\n");

    // Wait for lock files to exist, so know that data is in expected files and then remove output file and lock
    bool locksExist = false;

    while(!locksExist)
    {
        std::ifstream lock_a("./input_a.lock");
        std::ifstream lock_b("./input_b.lock");
        if(lock_a.is_open() && lock_b.is_open())
        {
            locksExist = true;
        }
        sleep(10);
    }
    
    printf("Removing output files.\n");
    remove( "./output_a.csv" );
    remove( "./output_a.lock" );
    remove( "./output_b.csv" );
    remove( "./output_b.lock" );

    string line_a;
    string line_b;

    printf("Parsing array from input csv files.\n");
    ifstream file_a ("./input_a.csv");
    ifstream file_b ("./input_b.csv");
    if (file_a.is_open() && file_b.is_open())
    {
        getline (file_a,line_a);
        printf("Parsing line: %s\n",line_a.c_str());
        parseIntsToArrayFromString(h_a, line_a, numElements);
        getline (file_b,line_b);
        printf("Parsing line: %s\n",line_b.c_str());
        parseIntsToArrayFromString(h_b, line_b, numElements);
    }
}

__host__ void parseIntsToArrayFromString(int * host_data, std::string line, int numElements)
{
    printf ("Splitting string \"%s\" into tokens\n",line.c_str());
    std::istringstream iss(line);
    std::string token;
    int i = 0;
    while (std::getline(iss, token, ','))
    {
        printf ("%s,",token.c_str());
        host_data[i] = std::stof(token);
        i++;
    }
    printf ("\n");
}

__host__ std::vector<std::string> split(const std::string& s, char delimiter)
{
   std::vector<std::string> tokens;
   std::string token;
   std::istringstream tokenStream(s);
   while (std::getline(tokenStream, token, delimiter))
   {
      tokens.push_back(token);
   }
   return tokens;
}

__host__ void performMultiCPUIteration()
{
    int numElements = 128;

    // Allocate host and device memory
    auto[h_a, h_b, h_c] = allocateHostMemory(numElements);
    auto[d_a, d_b] = allocateDeviceMemory(numElements);

    // Retrieve data from files
    retrieveDataFromFiles(h_a, h_b, numElements);

    // Copy data from host to the device
    copyFromHostToDevice(h_a, d_a, numElements);
    copyFromHostToDevice(h_b, d_b, numElements);

    // Execute kernel
    executeKernel(d_a, d_b, h_c, numElements);

    // Place data into files and synchronize the CUDA device
    placeDataToFiles(h_c, numElements);
    hipDeviceSynchronize();

    // Remove locks on input and lock on output, which should signal consumers to read
    remove( "./input_a.lock" );
    remove( "./input_b.lock" );

    // Create output lock files to signal python processes to print out data and generate input data
    fstream output_fstream_a;
    output_fstream_a.open("output_a.lock", std::ios_base::out);
    output_fstream_a.is_open();
    fstream output_fstream_b;
    output_fstream_b.open("output_b.lock", std::ios_base::out);
    output_fstream_b.is_open();


}

/*
 * Host main routine
 */
int main(void)
{
    int numIterations = 10; // This number can be changed
    for(int i = 0; i < numIterations; i++)
    {
        performMultiCPUIteration();
    }
    // Clean up device
    printf("Done\n");
    return 0;
}